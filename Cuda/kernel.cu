
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <fstream>

hipError_t addWithCuda(unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue, unsigned int size);
int checkSize(char* filename);
unsigned char* appendHeader(char* origin);
void readBMP(char* filename, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue);
void writeBMP(char* filename, char* origin, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue);
void get(unsigned char* color_from, unsigned char* color_to, int start, int end);
void set(unsigned char* color_from, unsigned char* color_to, int start, int end);

__global__ void addKernel(unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue)
{
	int i = threadIdx.x;
	char avg = (p_red[i] * 0.2126 + p_green[i] * 0.7152 + p_blue[i] * 0.0722);// (p_red[i] + p_green[i] + p_blue[i])/3;//(p_red[i] * 0.2126 + p_green[i] * 0.7152 + p_blue[i] * 0.0722);
	/*int ratio = 0;
	if (avg > 128)
	{
		avg -= (avg - 128) * 0.3;
	}
	else {
		avg += (128 - avg) * 0.3;
	}*/
	p_red[i] = avg;
	p_green[i] = avg;
	p_blue[i] = avg;
}

int main()
{
	int Block = 1024;
	int size = checkSize("C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\test.bmp");
	unsigned char* red = new unsigned char[size];
	unsigned char* green = new unsigned char[size];
	unsigned char* blue = new unsigned char[size];
	readBMP("C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\test.bmp", red, green, blue);
	int current_size = size;
	hipError_t cudaStatus = hipSuccess;
	while (current_size > 0)
	{
		if (current_size >= Block)
		{
			unsigned char* temp_red = new unsigned char[Block];
			unsigned char* temp_green = new unsigned char[Block];
			unsigned char* temp_blue = new unsigned char[Block];
			get(red, temp_red, size - current_size, size - current_size + Block);
			get(green, temp_green, size - current_size, size - current_size + Block);
			get(blue, temp_blue, size - current_size, size - current_size + Block);
			//cudaStatus = addWithCuda(temp_red, temp_green, temp_blue, Block);
			set(temp_red, red, size - current_size, size - current_size + Block);
			set(temp_green, green, size - current_size, size - current_size + Block);
			set(temp_blue, blue, size - current_size, size - current_size + Block);
			current_size -= Block;
		} else {
			unsigned char* temp_red = new unsigned char[current_size];
			unsigned char* temp_green = new unsigned char[current_size];
			unsigned char* temp_blue = new unsigned char[current_size];
			get(red, temp_red, size - current_size, size);
			get(green, temp_green, size - current_size, size);
			get(blue, temp_blue, size - current_size, size);
			//cudaStatus = addWithCuda(temp_red, temp_green, temp_blue, current_size);
			set(temp_red, red, size - current_size, size);
			set(temp_green, green, size - current_size, size);
			set(temp_blue, blue, size - current_size, size);
			current_size -= current_size;
		}
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
	}
    
    

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	writeBMP("C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\output.bmp", "C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\test.bmp", red, green, blue);
    return 0;
}

void get(unsigned char* color_from, unsigned char* color_to, int start, int end)
{
	int index = 0;
	for (int i = start; i < end; i++)
	{
		color_to[index] = color_from[i];
		index++;
	}
}

void set(unsigned char* color_from, unsigned char* color_to, int start, int end)
{
	int index = 0;
	for (int i = start; i < end; i++)
	{
		color_to[i] = color_from[index];
		index++;
	}
}

int checkSize(char* filename)
{
	FILE* f = fopen(filename, "rb");
	unsigned char info[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*)&info[18];
	int height = *(int*)&info[22];
	int size = width * height;
	fclose(f);
	return size;
}

unsigned char* appendHeader(char* origin)
{
	FILE* f = fopen(origin, "rb");
	unsigned char* info = new unsigned char [54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header
	fclose(f);
	return info;
}

void readBMP(char* filename, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue)
{
	int i;
	FILE* f = fopen(filename, "rb");
	unsigned char info[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*)&info[18];
	int height = *(int*)&info[22];

	int size = 3 * width * height;
	unsigned char* data = new unsigned char[size];
	fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
	fclose(f);

	int index = 0;

	for (i = 0; i < size; i += 3)
	{
		p_red[index] = data[i];
		p_green[index] = data[i + 1];
		p_blue[index] = data[i + 2];
		index++;
	}
}

void writeBMP(char* filename, char* originfile, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue)
{
	int i;
	unsigned char* info = appendHeader(originfile);

	FILE* f = fopen(filename, "w+b");

	int size = 3 * checkSize(originfile);
	unsigned char* data = new unsigned char[size];

	int index = 0;

	for (i = 0; i < size; i += 3)
	{
		data[i] = p_red[index];
		data[i + 1] = p_green[index];
		data[i + 2] = p_blue[index];
		index++;
	}
	fwrite(info, sizeof(unsigned char), 54, f);
	fwrite(data, sizeof(unsigned char), size, f); // read the rest of the data at once
	fclose(f);
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue, unsigned int size)
{
	unsigned char *dev_a = 0;
	unsigned char *dev_b = 0;
	unsigned char *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, p_red, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, p_green, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_c, p_blue, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_a, dev_b, dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	cudaStatus = hipMemcpy(p_red, dev_a, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(p_green, dev_b, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(p_blue, dev_c, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
	hipFree(dev_c);

    return cudaStatus;
}
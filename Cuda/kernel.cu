
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <fstream>

hipError_t addWithCuda(unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue, unsigned int size);
int checkSize(char* filename);
void appendHeader(char* filename, char* origin);
void readBMP(char* filename, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue);
void writeBMP(char* filename, char* origin, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue);
void get(unsigned char* color_from, unsigned char* color_to, int start, int end);
void set(unsigned char* color_from, unsigned char* color_to, int start, int end);

__global__ void addKernel(unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue)
{
	int i = threadIdx.x;
	/*int value_red = (((int)p_red[i]) <= 0) ?  (-1 * (int)p_red[i]) : (128 + (int)p_red[i]);
	int value_green = (((int)p_green[i]) <= 0) ? (-1 * (int)p_green[i]) : (128 + (int)p_green[i]);
	int value_blue = (((int)p_blue[i]) <= 0) ? (-1 * (int)p_blue[i]) : (128 + (int)p_blue[i]);
	char avg = (value_red * 299 + value_green * 587 + value_blue * 114) / 1000 - 128;*/
	int red_value = p_red[i];
	int green_value = p_green[i];
	int blue_value = p_blue[i];
	/*char avg = (red_value * 299 + green_value * 587 + blue_value * 114) / 1000;*/
	char avg = (red_value + green_value + blue_value) / 3;
	/*if (avg > 128)
	{
		avg -= (avg - 128) * 0.95;
	} else {
		avg += (128 - avg) * 0.95;
	}*/
	/*if (p_red[i] > 128)
	{
		p_red[i] -= (p_red[i] - 128) * 0.99;
	}
	else {
		p_red[i] += (128 - p_red[i]) * 0.99;
	}

	if (p_green[i] > 128)
	{
		p_green[i] -= (p_green[i] - 128) * 0.98;
	}
	else {
		p_green[i] += (128 - p_green[i]) * 0.98;
	}

	if (p_blue[i] > 128)
	{
		p_blue[i] -= (p_blue[i] - 128) * 0.95;
	}
	else {
		p_blue[i] += (128 - p_blue[i]) * 0.94;
	}*/

	p_red[i] = avg;
	p_green[i] = avg;
	p_blue[i] = avg;
}

int main()
{
	const int Block = 1024;
	int size = checkSize("C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\test.bmp");
	unsigned char* red = new unsigned char[size];
	unsigned char* green = new unsigned char[size];
	unsigned char* blue = new unsigned char[size];
	readBMP("C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\test.bmp", red, green, blue);
	int current_size = size;
	hipError_t cudaStatus;
	while (current_size > 0)
	{
		if (current_size >= Block)
		{
			unsigned char* temp_red = new unsigned char[Block];
			unsigned char* temp_green = new unsigned char[Block];
			unsigned char* temp_blue = new unsigned char[Block];
			get(red, temp_red, size - current_size, size - current_size + Block);
			get(green, temp_green, size - current_size, size - current_size + Block);
			get(blue, temp_blue, size - current_size, size - current_size + Block);
			//int value = temp_red[500];
			cudaStatus = addWithCuda(temp_red, temp_green, temp_blue, Block);
			set(temp_red, red, size - current_size, size - current_size + Block);
			set(temp_green, green, size - current_size, size - current_size + Block);
			set(temp_blue, blue, size - current_size, size - current_size + Block);
			current_size -= Block;
			delete temp_red;
			delete temp_green;
			delete temp_blue;
		} else {
			unsigned char* temp_red = new unsigned char[current_size];
			unsigned char* temp_green = new unsigned char[current_size];
			unsigned char* temp_blue = new unsigned char[current_size];
			get(red, temp_red, size - current_size, size);
			get(green, temp_green, size - current_size, size);
			get(blue, temp_blue, size - current_size, size);
			cudaStatus = addWithCuda(temp_red, temp_green, temp_blue, current_size);
			set(temp_red, red, size - current_size, size);
			set(temp_green, green, size - current_size, size);
			set(temp_blue, blue, size - current_size, size);
			current_size -= current_size;
			delete temp_red;
			delete temp_green;
			delete temp_blue;
		}
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
	}
    
    

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	writeBMP("C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\output.bmp", "C:\\Users\\mateu\\source\\repos\\cuda\\x64\\Debug\\test.bmp", red, green, blue);
	delete red;
	delete green;
	delete blue;
    return 0;
}

void get(unsigned char* color_from, unsigned char* color_to, int start, int end)
{
	int index = 0;
	for (int i = start; i < end - start; i++)
	{
		color_to[index] = color_from[i];
		index++;
	}
}

void set(unsigned char* color_from, unsigned char* color_to, int start, int end)
{
	int index = 0;
	for (int i = start; i < end - start; i++)
	{
		color_to[i] = color_from[index];
		index++;
	}
}

int checkSize(char* filename)
{
	FILE* f = fopen(filename, "rb");
	unsigned char info[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*)&info[18];
	int height = *(int*)&info[22];
	int size = width * height;
	fclose(f);
	return size;
}

void appendHeader(char* filename, char* origin)
{
	FILE* f = fopen(origin, "rb");
	unsigned char* info = new unsigned char[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header
	fclose(f);
	f = fopen(filename, "wb");
	fwrite(info, sizeof(unsigned char), 54, f);
	fclose(f);
	delete info;
}

void readBMP(char* filename, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue)
{
	int i;
	FILE* f = fopen(filename, "rb");
	unsigned char info[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*)&info[18];
	int height = *(int*)&info[22];

	int size = 3 * width * height;
	unsigned char* data = new unsigned char[size];
	fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
	fclose(f);

	int index = 0;

	for (i = 0; i < size; i += 3)
	{
		int red = data[i];
		int green = data[i + 1];
		int blue = data[i + 2];
		p_red[index] = red;
		p_green[index] = green;
		p_blue[index] = blue;
		index++;
	}
	delete data;
}

void writeBMP(char* filename, char* originfile, unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue)
{
	int i;
	appendHeader(filename, originfile);

	FILE* f = fopen(filename, "a+b");

	int size = 3 * checkSize(originfile);
	unsigned char* data = new unsigned char[size];

	int index = 0;

	for (i = 0; i < size; i += 3)
	{
		data[i] = p_red[index];
		data[i + 1] = p_green[index];
		data[i + 2] = p_blue[index];
		index++;
	}

	fwrite(data, sizeof(unsigned char), size, f); // read the rest of the data at once
	fclose(f);
	delete data;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(unsigned char* p_red, unsigned char* p_green, unsigned char* p_blue, unsigned int size)
{
	unsigned char *dev_a = 0;
	unsigned char *dev_b = 0;
	unsigned char *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, p_red, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, p_green, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_c, p_blue, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_a, dev_b, dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	cudaStatus = hipMemcpy(p_red, dev_a, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(p_green, dev_b, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(p_blue, dev_c, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
	hipFree(dev_c);

    return cudaStatus;
}